#include "Material.h"

CUDA_DEVICE bool Lambertian::scatter(const Ray& inRay, const HitResult& hitResult, Vector3Df& attenuation, Ray& scattered, hiprandState* randState) const {
    //auto scatterDirection = Utils::randomUnitVector(randState);                                         // Diffuse1
    auto scatterDirection = hitResult.normal + Utils::randomUnitVector(randState);                      // Diffuse2
    //auto scatterDirection = Utils::randomHemiSphere(hitResult.normal, randState);                       // Diffuse3
    //auto scatterDirection = hitResult.normal + Utils::randomHemiSphere(hitResult.normal, randState);    // Diffuse4
    //auto scatterDirection = hitResult.normal + Utils::randomInUnitSphere(randState);                    // Diffuse5
    // Catch degenerate scatter direction
    // If the random unit vector we generate is exactly opposite the normal vector, 
    // the two will sum to zero, which will result in a zero scatter direction vector. 
    // This leads to bad scenarios later on (infinities and NaNs),
    if (Utils::nearZero(scatterDirection)) {
        scatterDirection = hitResult.normal;
    }
    scattered = Ray(inRay.at(hitResult.t), normalize(scatterDirection), inRay.time);
    attenuation = albedo;
    return true;
}