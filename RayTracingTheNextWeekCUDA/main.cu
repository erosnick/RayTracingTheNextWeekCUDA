#include "hip/hip_runtime.h"
﻿
#include "main.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include "Utils.h"
#include "GPUTimer.h"
#include "Sphere.h"
#include "Plane.h"
#include "Triangle.h"
#include "Mesh.h"
#include "Cube.h"
#include "YAML.h"
#include "ModelLoader.h"

#include <yaml-cpp/yaml.h>
#include <cstdio>
#include <algorithm>

template<typename T>
T* createObjectPtr() {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T*)));
    return object;
}

template<typename T>
T* createObjectArray(int32_t numObjects) {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T) * numObjects));
    return object;
}

template<typename T>
T* createObjectPtrArray(int32_t numObjects) {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T) * numObjects));
    return object;
}

template<typename T>
void deleteObject(T* object) {
    gpuErrorCheck(hipFree(object));
}

template<typename T>
CUDA_GLOBAL void deleteDeviceObject(T** object) {
    delete (*object);
}

template<typename T>
CUDA_GLOBAL void deleteDeviceObjectArray(T** object, int32_t count) {
    for (auto i = 0; i < count; i++) {
        delete *(object + i);
    }
}

constexpr auto BOUNCES = 4;

constexpr auto OBJECTS = 7;
constexpr auto MATERIALS = 6;
CUDA_CONSTANT Hitable* constantObjects[OBJECTS];
CUDA_CONSTANT Material* constantMaterials[MATERIALS];

CUDA_DEVICE bool hit(const Ray& ray, Float tMin, Float tMax, HitResult& hitResult) {
    HitResult tempHitResult;
    bool bHitAnything = false;
    Float closestSoFar = tMax;
    for (auto& object : constantObjects) {
        // Empty hit call costs ~130ms
        if (object->hit(ray, tMin, closestSoFar, tempHitResult)) {
            bHitAnything = true;
            closestSoFar = tempHitResult.t;
            hitResult = tempHitResult;
        }
    }

    return bHitAnything;
}

using ScatterFunction = bool (*)(const Ray& inRay, const HitResult& hitResult, Float3& attenuation, Ray& scattered, hiprandState* randState);

CUDA_DEVICE ScatterFunction scatterFunction;

CUDA_DEVICE bool lambertianScatter(const Ray& inRay, const HitResult& hitResult, Float3& attenuation, Ray& scattered, hiprandState* randState) {
    auto scatterDirection = hitResult.normal + Utils::randomUnitVector(randState);
    if (Utils::nearZero(scatterDirection)) {
        scatterDirection = hitResult.normal;
    }
    scattered = Ray(inRay.at(hitResult.t), normalize(scatterDirection), inRay.time);
    attenuation = make_float3(1.0f, 1.0f, 1.0f);
    return true;
}

CUDA_DEVICE bool emissionScatter(const Ray& inRay, const HitResult& hitResult, Float3& attenuation, Ray& scattered, hiprandState* randState) {
    attenuation = make_float3(1.0f, 1.0f, 1.0f) * 5.0f;
    return false;
}

CUDA_DEVICE Float3 rayColor(const Ray& ray, hiprandState* randState) {
    Ray currentRay = ray;
    auto currentAttenuation = make_float3(1.0f, 1.0f, 1.0f);
    for (auto i = 0; i < BOUNCES; i++) {
        HitResult hitResult;
        // Smaller tMin will has a impact on performance
        if (hit(currentRay, Math::epsilon, Math::infinity, hitResult)) {
            Float3 attenuation;
            Ray scattered;
            // Bounces 4 Samples 100 18ms
            // Bounces 4 Samples 100 33ms(Empty scatter function body)
            if (constantMaterials[hitResult.materialId]->scatter(currentRay, hitResult, attenuation, scattered, randState)) {
                currentAttenuation *= attenuation;
                currentRay = scattered;
            }
            else {
                return currentAttenuation * attenuation;
            }
        }
        // If no intersection in the first bounce, just return background color
        // otherwise return currentAttenuation * background color
        else {
            auto unitDirection = normalize(currentRay.direction);
            auto t = 0.5f * (unitDirection.y + 1.0f);
            auto background = lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), t);
            return currentAttenuation * background;
        }
    }
    // exceeded recursion
    return make_float3(0.0f, 0.0f, 0.0f);
}

//CUDA_DEVICE Float3 rayColor(const Ray& ray, hiprandState* randState, Sphere* spheres, int32_t depth) {
//    if (depth == 0) {
//        // exceeded recursion
//        return make_float3(0.0f, 0.0f, 0.0f);
//    }
//    HitResult hitResult;
//    // Smaller tMin will has a impact on performance
//    if (hit(ray, Math::epsilon, Math::infinity, hitResult, spheres)) {
//        Float3 attenuation;
//        Ray rayScattered;
//        if (hitResult.material->scatter(ray, hitResult, attenuation, rayScattered, randState)) {
//            return attenuation * rayColor(rayScattered, randState, spheres, depth - 1);
//        }
//        else {
//            return currentAttenuation * attenuation;
//        }
//    }
//
//    auto unitDirection = normalize(ray.direction);
//    auto t = 0.5f * (unitDirection.y + 1.0f);
//    auto background = lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), t);
//    return background;
//}

CUDA_GLOBAL void renderInit(int32_t width, int32_t height, hiprandState* randState) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto index = y * width + x;

    if (index < (width * height)) {
        //Each thread gets same seed, a different sequence number, no offset
        hiprand_init(1984, index, 0, &randState[index]);
    }
}

//CUDA_GLOBAL void render(Canvas canvas, Camera camera, hiprandState* randStates, Sphere* spheres) {
//    auto x = threadIdx.x + blockDim.x * blockIdx.x;
//    auto y = threadIdx.y + blockDim.y * blockIdx.y;
//    auto width = canvas.getWidth();
//    auto height = canvas.getHeight();
//    constexpr auto samplesPerPixel = 1;
//    constexpr auto maxDepth = 5;
//    auto index = y * width + x;
//
//    if (index < (width * height)) {
//        auto color = make_float3(0.0f, 0.0f, 0.0f);
//        auto localRandState = randStates[index];
//        for (auto i = 0; i < samplesPerPixel; i++) {
//
//            auto rx = hiprand_uniform(&localRandState);
//            auto ry = hiprand_uniform(&localRandState);
//
//            auto dx = Float(x + rx) / (width - 1);
//            auto dy = Float(y + ry) / (height - 1);
//
//            auto ray = camera.getRay(dx, dy);
//            color += rayColor(ray, &localRandState, spheres);
//        }
//        // Very important!!!
//        randStates[index] = localRandState;
//        canvas.writePixel(index, color / samplesPerPixel);
//    }
//}

CUDA_GLOBAL void renderKernel(Canvas* canvas, Camera* camera, hiprandState* randStates, int32_t* counter) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto width = canvas->getWidth();
    auto height = canvas->getHeight();
#ifdef GPU_REALTIME
    constexpr auto samplesPerPixel = 1;
#else
    constexpr auto samplesPerPixel = 8;
#endif // GPU_REALTIME

    constexpr auto maxDepth = 5;
    auto index = y * width + x;

    if (index < (width * height)) {
        auto color = make_float3(0.0f, 0.0f, 0.0f);
        auto localRandState = randStates[index];
        for (auto i = 0; i < samplesPerPixel; i++) {

            auto rx = hiprand_uniform(&localRandState);
            auto ry = hiprand_uniform(&localRandState);

            auto dx = Float(x + rx) / (width - 1);
            auto dy = Float(y + ry) / (height - 1);

            auto ray = camera->getRay(dx, dy, &localRandState);
            color += rayColor(ray, &localRandState);
        }
        // Very important!!!
        randStates[index] = localRandState; 
#ifdef GPU_REALTIME
        canvas->accumulatePixel(index, color);
#else
        canvas->writePixel(index, color / samplesPerPixel);

        auto tenPercent = (width * height) / 10;

        auto old = atomicAdd(counter, 1);

        if ((old + 1) > 0 && (old + 1) % tenPercent == 0) {
            printf("Complete: %.2f%%\n", (float((old + 1) * 100) / (width * height)));
        }
#endif // GPU_REALTIME
    }
}

CUDA_GLOBAL void createLambertianMaterialKernel(Material** material, int32_t index, Float3 albedo, Float absorb = 1.0f) {
    *(material + index) = new Lambertian(index, albedo, absorb);
}

void createLambertianMaterial(Material** material, int32_t index, Float3 albedo, Float absorb = 1.0f) {
    createLambertianMaterialKernel<<<1, 1>>>(material, index, albedo, absorb);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createEmissionMaterialKernel(Material** material, int32_t index, Float3 albedo, Float intensity = 1.0f) {
    *(material + index) = new Emission(index, albedo, intensity);
}

void createEmissionMaterial(Material** material, int32_t index, Float3 albedo, Float intensity = 1.0f) {
    createEmissionMaterialKernel<<<1, 1>>>(material, index, albedo, intensity);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createMetalMaterialKernel(Material** material, int32_t index, Float3 albedo, Float fuzz = 1.0f) {
    *(material + index) = new Metal(index, albedo, fuzz);
}

void createMetalMaterial(Material** material, int32_t index, Float3 albedo, Float fuzz = 1.0f) {
    createMetalMaterialKernel<<<1, 1>>>(material, index, albedo, fuzz);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createDieletricMaterialKernel(Material** material, int32_t index, Float indexOfRefraction = 1.5f) {
    *(material + index) = new Dieletric(index, indexOfRefraction);
}

void createDieletricMaterial(Material** material, int32_t index, Float indexOfRefraction = 1.5f) {
    createDieletricMaterialKernel<<<1, 1>>>(material, index, indexOfRefraction);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void clearBackBuffers(Canvas* canvas) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto width = canvas->getWidth();
    auto height = canvas->getHeight();

    auto index = y * width + x;

    if (index < (width * height)) {
        canvas->clearPixel(index);
    }
}

CUDA_GLOBAL void createSphereKernel(Hitable** sphere, int32_t index, Float3 center, Float radius, Material* material, bool bShading) {
    *(sphere + index) = new Sphere(center, radius, material, bShading);
}

void createSphere(Hitable** sphere, int32_t index, Float3 center, Float radius, Material* material, bool bShading = true) {
    createSphereKernel<<<1, 1>>>(sphere, index, center, radius, material, bShading);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createMovingSphereKernel(Hitable** sphere, int32_t index, Float3 center0, Float3 center1, Float time0, Float time1, Float radius, Material* material) {
    *(sphere + index) = new MovingSphere(center0, center1, time0, time1, radius, material);
}

void createMovingSphere(Hitable** sphere, int32_t index, Float3 center0, Float3 center1, Float time0, Float time1, Float radius, Material* material) {
    createMovingSphereKernel<<<1, 1>>>(sphere, index, center0, center1, time0, time1, radius, material);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createPlaneKernel(Hitable** plane , int32_t index, Float3 position, Float3 normal, Float3 extend, Material* material, PlaneOrientation orientation) {
    *(plane + index) = new Plane(position, normal, extend, material, orientation);
}

void createPlane(Hitable** plane, int32_t index, const Float3& position, const Float3& normal, const Float3& extend, Material* material, PlaneOrientation orientation) {
    createPlaneKernel<<<1, 1>>>(plane, index, position, normal, extend, material, orientation);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createTriangleKernel(Hitable** triangle, int32_t index, Float3 v0, Float3 v1, Float3 v2, Material* material) {
    *(triangle + index) = new Triangle(v0, v1, v2, material);
}

void createTriangle(Hitable** triangle, int32_t index, const Float3& v0, const Float3& v1, const Float3& v2, Material* material) {
    createTriangleKernel<<<1, 1>>>(triangle, index, v0, v1, v2, material);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createMeshKernel(Hitable** mesh, int32_t index, Hitable** triangles, int32_t triangleCount, Material* material) {
    *(mesh + index) = new Mesh(triangles, triangleCount, material);
}

void createMesh(Hitable** triangle, int32_t index, Hitable** triangles, int32_t triangleCount, Material* material) {
    createMeshKernel<<<1, 1>>>(triangle, index, triangles, triangleCount, material);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createCubeKernel(Hitable** cube, int32_t index, Float3 position, Hitable** faces, Material* material) {
    *(cube + index) = new Cube(position, faces, material);
}

void createCube(Hitable** triangle, int32_t index, Float3 position, Hitable** faces, Material* material) {
    createCubeKernel<<<1, 1>>>(triangle, index, position, faces, material);
    gpuErrorCheck(hipDeviceSynchronize());
}

#define RESOLUTION 1

#if RESOLUTION == 0
int32_t width = 512;
int32_t height = 384;
#elif RESOLUTION == 1
int32_t width = 512;
int32_t height = 512;
#elif RESOLUTION == 2
int32_t width = 1024;
int32_t height = 1024;
#elif RESOLUTION == 3
int32_t width = 1024;
int32_t height = 576;
#elif RESOLUTION == 4
int32_t width = 1280;
int32_t height = 720;
#elif RESOLUTION == 5
int32_t width = 1920;
int32_t height = 1080;
#elif RESOLUTION == 6
int32_t width = 64;
int32_t height = 36;
#endif

#define SCENE 1

Canvas* canvas = nullptr;
Camera* camera = nullptr;
Hitable** spheres = nullptr;
Hitable** triangles = nullptr;
int32_t triangleCount = 0;
//std::vector<Material**> materials(MATERIALS);
Material** materials = nullptr;
hiprandState* randStates = nullptr;
std::shared_ptr<ImageData> imageData = nullptr;

dim3 blockSize(32, 32);
dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
              (height + blockSize.y - 1) / blockSize.y);

void loadScene(const std::string& path) {
    YAML::Node config = YAML::LoadFile(path);

    printf("name:%s\n", config["name"].as<std::string>().c_str());
    printf("sex:%s\n", config["sex"].as<std::string>().c_str());
    printf("age:%d\n", config["age"].as<int>());

    for (auto iterator = config["skills"].begin(); iterator != config["skills"].end(); iterator++) {
        printf("%s\n", iterator->first.as<std::string>().c_str());
    }
}

void initialize(int32_t width, int32_t height) {
    //Canvas canvas(width, height);
    Utils::reportGPUUsageInfo();
    canvas = createObjectPtr<Canvas>();
    canvas->initialize(width, height);

    //Camera camera(make_float3(-2.0f, 2.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 20.0f);
    camera = createObjectPtr<Camera>();
    //camera->initialize(make_float3(-2.0f, 2.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 20.0f);
    //camera->initialize(make_float3(0.0f, 1.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 90.0f);

    //auto eye = make_float3(3.0f, 3.0f, 5.0f);
    //auto center = make_float3(0.0f, 0.0f, -1.0f);
    //auto up = make_float3(0.0f, 1.0f, 0.0f);
    //auto focusDistance = length(center - eye);
    //camera->initialize(eye, center, up, Float(width) / height, 20.0f, 2.0f, focusDistance);

    // If the distance between object and camera equals to focus lens
    // then the object is in focus
    //auto eye = position(3.0f, 3.0f, 5.0f);
    //auto center = position(0.0f, 0.0f, -1.0f);
    //auto up = position(0.0f, 1.0f, 0.0f);
    //auto focusDistance = length(center - eye);
    //camera->initialize(eye, center, up, Float(width) / height, 20.0f, 2.0f, focusDistance);

    //loadScene("./resources/scenes/test.yaml");

    //for (auto& material : materials) {
    //    material = createObjectPtr<Material*>();
    //}

    materials = createObjectPtrArray<Material*>(MATERIALS);

    spheres = createObjectPtrArray<Hitable*>(OBJECTS);

#if SCENE == 0
    // If the distance between object and camera equals to focus lens
    // then the object is in focus
    auto eye = point(3.0f, 3.0f, 5.0f);
    auto center = point(0.0f, 0.0f, -1.0f);
    auto up = point(0.0f, 1.0f, 0.0f);
    auto focusDistance = length(center - eye);
    auto aperture = 0.0f;
    camera->initialize(eye, center, up, Float(width) / height, 20.0f, aperture, focusDistance, 0.0f, 1.0f);

    // Scene1 Defocus Blur
    createDieletricMaterial(materials, 0, 1.5f);
    createDieletricMaterial(materials, 1, 1.5f);
    createLambertianMaterial(materials, 2, make_float3(0.1f, 0.2f, 0.5f));
    //createDieletricMaterial<<<1, 1>>>(materials[3], 1.5f);
    createMetalMaterial(materials, 3, make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    //createLambertianMaterial<<<1, 1>>>(materials[4], make_float3(0.8f, 0.8f, 0.0f));
    createMetalMaterial(materials, 4, make_float3(0.5f, 0.7f, 1.0f), 0.0f);

    auto center1 = point(0.0f, 0.5f, 0.0f);

    //createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f}, 0.5f, *(materials[0]));
    //createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    //createSphere(spheres, 2, {  0.0f, 0.0f, -1.0f },  0.5f, *(materials[2]));
    //createSphere(spheres, 3, {  1.0f, 0.0f, -1.0f },  0.5f, *(materials[3]));
    //createSphere(spheres, 4, {  0.0f, -100.5f, -1.0f }, 100.0f, *(materials[4]));
    createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f}, 0.5f, materials[0]);
    createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, materials[1], false);
    createMovingSphere(spheres, 2, {  0.0f, 0.0f, -1.0f }, { 0.0f, 0.5f, -1.0f }, 0.0f, 1.0f, 0.5f, materials[2]);
    createSphere(spheres, 3, {  1.0f, 0.0f, -1.0f },  0.5f, materials[3]);
    createSphere(spheres, 4, {  0.0f, -100.5f, -1.0f }, 100.0f, materials[4]);
#elif SCENE == 1
    // If the distance between object and camera equals to focus lens
    // then the object is in focus
    auto eye = point(0.0f, 0.0f, 10.0f);
    auto center = point(0.0f, 0.0f, -1.0f);
    auto up = point(0.0f, 1.0f, 0.0f);
    auto focusDistance = length(center - eye);
    auto aperture = 0.0f;
    camera->initialize(eye, center, up, Float(width) / height, 20.0f, aperture, focusDistance, 0.0f, 1.0f);

    // Scene1 Defocus Blur
    //createDieletricMaterial(materials, 0, 1.5f);
    //createDieletricMaterial(materials, 1, 1.5f);
    //createLambertianMaterial(materials, 2, make_float3(0.1f, 0.2f, 0.5f));
    createLambertianMaterial(materials, 0, make_float3(1.0f, 0.0f, 0.0f));
    createLambertianMaterial(materials, 1, make_float3(0.0f, 1.0f, 0.0f));
    createLambertianMaterial(materials, 2, make_float3(0.0f, 0.0f, 1.0f));
    createLambertianMaterial(materials, 3, make_float3(1.0f, 1.0f, 1.0f));
    createLambertianMaterial(materials, 4, make_float3(0.8f, 0.8f, 0.9f));
    //createDieletricMaterial<<<1, 1>>>(materials[3], 1.5f);
    //createMetalMaterial(materials, 3, make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    //createLambertianMaterial(materials, 4, make_float3(0.8f, 0.8f, 0.0f));
    //createMetalMaterial(materials, 4, make_float3(0.5f, 0.7f, 1.0f), 0.0f);

    auto center1 = point(0.0f, 0.5f, 0.0f);

    //createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f}, 0.5f, *(materials[0]));
    //createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    //createSphere(spheres, 2, {  0.0f, 0.0f, -1.0f },  0.5f, *(materials[2]));
    //createSphere(spheres, 3, {  1.0f, 0.0f, -1.0f },  0.5f, *(materials[3]));
    //createSphere(spheres, 4, {  0.0f, -100.5f, -1.0f }, 100.0f, *(materials[4]));

    //auto model = loadModel("./resources/models/bunny/bunny.obj");
    //auto model = loadModel("./resources/models/cube/cube.obj");
    //auto model = loadModel("./resources/models/plane/plane.obj");
    //auto model = loadModel("./resources/models/test/test.obj");
    auto model = loadModel("./resources/models/suzanne/suzanne.obj");

    std::vector<Float> positionX;
    std::vector<Float> positionY;
    std::vector<Float> positionZ;

    for (const auto& vertex : model) {
        positionX.push_back(vertex.x);
        positionY.push_back(vertex.y);
        positionZ.push_back(vertex.z);
    }

    std::sort(positionX.begin(), positionX.end());
    std::sort(positionY.begin(), positionY.end());
    std::sort(positionZ.begin(), positionZ.end());

    Float3 minAABB = { positionX[0], positionY[0], positionZ[0] };
    Float3 maxAABB = { positionX[positionX.size()- 1],  positionY[positionY.size() - 1], positionZ[positionZ.size() - 1] };

    Float3 extendAABB = (maxAABB - minAABB) * 0.5f;

    Float3 centerAABB = (minAABB + maxAABB) * 0.5f;

    triangleCount = model.size() / 3;
    triangles = createObjectPtrArray<Hitable*>(triangleCount);

    //for (auto i = 0; i < triangleCount; i++) {
    //    createTriangle(triangles, i, model[i * 3], model[i * 3 + 1], model[i * 3 + 2], materials[3]);
    //}

    //createMesh(spheres, 0, triangles, triangleCount, materials[0]);

    createSphere(spheres, 0, { 0.0f, -102.0f, -1.0f }, 100.0f, materials[4]);

    createPlane(spheres, 1, { centerAABB.x - extendAABB.x, centerAABB.y, centerAABB.z }, { -1.0f, 0.0f, 0.0f }, extendAABB, materials[0], PlaneOrientation::YZ);   // Left
    createPlane(spheres, 2, { centerAABB.x + extendAABB.x, centerAABB.y, centerAABB.z }, {  1.0f, 0.0f, 0.0f }, extendAABB, materials[0], PlaneOrientation::YZ);   // Right
    createPlane(spheres, 3, { centerAABB.x, centerAABB.y + extendAABB.y, centerAABB.z }, { 0.0f,  1.0f, 0.0f }, extendAABB, materials[0], PlaneOrientation::XZ);   // Top
    createPlane(spheres, 4, { centerAABB.x, centerAABB.y - extendAABB.y, centerAABB.z }, { 0.0f, -1.0f, 0.0f }, extendAABB, materials[0], PlaneOrientation::XZ);   // Bottom
    createPlane(spheres, 5, { centerAABB.x, centerAABB.y, centerAABB.z + extendAABB.z }, { 0.0f, 0.0f,  1.0f }, extendAABB, materials[0], PlaneOrientation::XY);   // Front
    createPlane(spheres, 6, { centerAABB.x, centerAABB.y, centerAABB.z - extendAABB.z }, { 0.0f, 0.0f, -1.0f }, extendAABB, materials[0], PlaneOrientation::XY);   // Back

#elif SCENE == 2
    // If the distance between object and camera equals to focus lens
    // then the object is in focus
    YAML::Node scene = YAML::LoadFile("./resources/scenes/cornellbox.yaml");

    auto eye = scene["camera"]["eye"].as<Float3>();
    auto center = scene["camera"]["center"].as<Float3>();
    auto up = scene["camera"]["up"].as<Float3>();
    auto focusDistance = length(center - eye);
    auto aperture = scene["camera"]["aperture"].as<Float>();
    auto fov = scene["camera"]["fov"].as<Float>();
    camera->initialize(eye, center, up, Float(width) / height, fov, aperture, focusDistance, 0.0f, 1.0f);

    //// Scene1 Defocus Blur
    ////createLambertianMaterial(materials[0], make_float3(0.0f, 1.0f, 0.0f));
    ////createLambertianMaterial(materials[1], make_float3(1.0f, 0.0f, 0.0f));
    //createLambertianMaterial(materials, 2, make_float3(1.0f, 1.0f, 1.0f));
    //createDieletricMaterial(materials, 3, 1.5f);
    //createMetalMaterial(materials, 4, make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    //createMetalMaterial(materials, 4, make_float3(1.0f, 1.0f, 1.0f), 0.0f);
    //createLambertianMaterial(materials, 5, make_float3(0.8f, 0.8f, 0.0f));
    //createMetalMaterial(materials, 6, make_float3(0.5f, 0.7f, 1.0f), 0.0f);

    ////createLambertianMaterial(materials[5], make_float3(1.0f, 1.0f, 1.0f));
    //createDieletricMaterial(materials, 7, 1.5f);
    //createDieletricMaterial(materials, 8, 1.5f);
    //createEmissionMaterial(materials, 5, make_float3(1.0f, 1.0f, 1.0f), 10.0f);

    auto objects = scene["objects"];

    for (auto i = 0; i < OBJECTS; i++) {
        // 场景的构成是objects是几何体数组
        // 数组的元素是Map，其中又包含若干几何体属性Map
        // objects:
        //    -sphere : # Left
        //        type : 0
        //        center :
        //          - -1000.5
        //          -  0.0
        //          -  0.0
        //        radius : 1000
        //        materialId : 1
        //        material :
        //        type : 0
        //        albedo :
        //          - 0.75
        //          - 0.25
        //          - 0.25
        // 表示Map中第一个元素的迭代器，这里sphere是一个Map
        // 这里的key就是字符串"sphere"
        auto iterator = objects[i].begin();
        auto key = iterator->first.as<std::string>();

        auto object = objects[i][key];

        auto materialType = static_cast<MaterialType>(object["material"]["type"].as<uint8_t>());

        iterator = object.begin();

        auto materialId = object["materialId"].as<uint32_t>();

        switch (materialType) {
            case MaterialType::Lambertian: {
                auto albedo = object["material"]["albedo"].as<Float3>();

                if ((materials[materialId]) == nullptr) {
                    createLambertianMaterial(materials, materialId, albedo);
                }
            }
        
            break;
            case MaterialType::Dieletric: {
                auto indexOfRefraction = object["material"]["indexOfRefraction"].as<Float>();

                if ((materials[materialId]) == nullptr) {
                    createDieletricMaterial(materials, materialId, indexOfRefraction);
                }
            }
                                        
            break;
            case MaterialType::Metal: {
                auto albedo = object["material"]["albedo"].as<Float3>();
                auto fuzz = object["material"]["fuzz"].as<Float>();

                if ((materials[materialId]) == nullptr) {
                    createMetalMaterial(materials, materialId, albedo, fuzz);
                }
            }

            break;
            case MaterialType::Emission: {
                auto albedo = object["material"]["albedo"].as<Float3>();
                auto intensity = object["material"]["intensity"].as<Float>();

                if ((materials[materialId]) == nullptr) {
                    createEmissionMaterial(materials, materialId, albedo, intensity);
                }
            }
        }

        auto primitiveType = static_cast<PrimitiveType>(iterator->second.as<uint8_t>());

        switch (primitiveType) {
            case PrimitiveType::Sphere: {
                auto center = object["center"].as<Float3>();
                auto radius = object["radius"].as<Float>();

                createSphere(spheres, i, center, radius, materials[materialId]);
            }
            
            break;
            case PrimitiveType::Plane: {
                auto position = object["position"].as<Float3>();
                auto normal = object["normal"].as<Float3>();
                auto extend = object["extend"].as<Float3>();

                createPlane(spheres, i, position, normal, extend, materials[materialId]);
            }

            break;
            case PrimitiveType::Triangle: {
                auto v0 = object["v0"].as<Float3>();
                auto v1 = object["v1"].as<Float3>();
                auto v2 = object["v2"].as<Float3>();

                createTriangle(spheres, i, v0, v1, v2, materials[materialId]);
            }

            break;
        }
    }

    //// If the distance between object and camera equals to focus lens
    //// then the object is in focus
    //auto eye = point(0.0f, 0.0f, 1.5f);
    //auto center = point(0.0f, 0.0f, -1.0f);
    //auto up = point(0.0f, 1.0f, 0.0f);
    //auto focusDistance = length(center - eye);
    //auto aperture = 0.0f;
    //camera->initialize(eye, center, up, Float(width) / height, 45.0f, aperture, focusDistance, 0.0f, 1.0f);

    //// Scene1 Defocus Blur
    //createDieletricMaterial(materials[0], 1.5f);
    //createDieletricMaterial(materials[1], 1.5f);
    //createLambertianMaterial(materials[2], make_float3(0.1f, 0.2f, 0.5f));
    ////createDieletricMaterial<<<1, 1>>>(materials[3], 1.5f);
    //createMetalMaterial(materials[3], make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    ////createLambertianMaterial<<<1, 1>>>(materials[4], make_float3(0.8f, 0.8f, 0.0f));
    //createMetalMaterial(materials[4], make_float3(0.5f, 0.7f, 1.0f), 0.0f);
    //createLambertianMaterial(materials, 5, make_float3(1.0f, 1.0f, 1.0f));

    ////createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f}, 0.5f, *(materials[0]));
    ////createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    ////createSphere(spheres, 2, {  0.0f, 0.0f, -1.0f },  0.5f, *(materials[2]));
    ////createSphere(spheres, 3, {  1.0f, 0.0f, -1.0f },  0.5f, *(materials[3]));
    ////createSphere(spheres, 4, {  0.0f, -100.5f, -1.0f }, 100.0f, *(materials[4]));
    ////createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f }, 0.5f, *(materials[5]));
    ////createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    ////createSphere(spheres, 2, { 0.0f, 0.0f, -1.0f }, 0.5f, *(materials[5]));
    //createSphere(spheres, 0, {  0.25f, -0.325f, -0.125f }, 0.175f, *(materials[5]));
    //createSphere(spheres, 1, { -0.25f, -0.325f, -0.25f }, 0.175f, * (materials[5]));
    //createSphere(spheres, 2, {  0.0f, -1000.5f, 0.0f }, 1000.0f, *(materials[5]));
#else
    auto eye = point(13.0f, 2.0f, 3.0f);
    auto center = point(0.0f, 0.0f, 0.0f);
    auto up = point(0.0f, 1.0f, 0.0f);
    auto focusDistance = 10.0f;
    auto aperture = 0.1f;
    camera->initialize(eye, center, up, Float(width) / height, 20.0f, aperture, focusDistance, 0.0f, 1.0f);

    // Scene2 Final
    for (auto a = -11; a < 11; a++) {
        for (auto b = -11; b < 11; b++) {
            auto index = (a + 11) * 22 + (b + 11);
            auto chooseMaterial = Utils::randomFloat();

            auto center = point(a + 0.9f * Utils::randomFloat(), 0.2f, b + 0.9f * Utils::randomFloat());

            if (length(center - point(4.0f, 0.2f, 0.0f)) > 0.9f) {
                if (chooseMaterial < 0.8f) {
                    // Diffuse
                    auto albedo = Color::random() * Color::random();
                    createLambertianMaterial(materials[index], albedo);
                    auto center1 = center + point(0.0f, Utils::randomFloat(0.0f, 0.5f), 0.0f);
                    createMovingSphere(spheres, index, center, center1, 0.0f, 1.0f, 0.2f, *(materials[index]));
                    //createSphere(spheres, index, center, 0.2f, *(materials[index]));

                }
                else if (chooseMaterial < 0.95f) {
                    // Metal
                    auto albedo = Color::random(0.5f, 1.0f);
                    auto fuzz = Utils::randomFloat(0.0f, 0.5f);
                    createMetalMaterial(materials[index], albedo, fuzz);
                    createSphere(spheres, index, center, 0.2f, *(materials[index]));
                }
                else {
                    // Glass
                    createDieletricMaterial(materials[index], 1.5f);
                    createSphere(spheres, index, center, 0.2f, *(materials[index]));
                }
            }
            else {
                auto albedo = Color::random() * Color::random();
                createLambertianMaterial(materials[index], albedo);
                createSphere(spheres, index, center, 0.2f, *(materials[index]));
            }
        }
    }

    createLambertianMaterial(materials[484], color(0.5f, 0.5f, 0.5f));
    createDieletricMaterial(materials[485], 1.5f);
    createLambertianMaterial(materials[486], color(0.4f, 0.2f, 0.1f), 1.0f);
    createMetalMaterial(materials[487], color(0.7f, 0.6f, 0.5f), 0.0f);

    createSphere(spheres, 484, point( 0.0f, -1000.0,  0.0f), 1000.0f, *(materials[484]));
    createSphere(spheres, 485, point( 0.0f,     1.0f, 0.0f),    1.0f, *(materials[485]));
    createSphere(spheres, 486, point(-4.0f,     1.0f, 0.0f),    1.0f, *(materials[486]));
    createSphere(spheres, 487, point( 4.0f,     1.0f, 0.0f),    1.0f, *(materials[487]));

#endif
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(constantObjects), spheres, sizeof(Hitable*) * OBJECTS));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(constantMaterials), materials, sizeof(Material*) * MATERIALS));

    auto pixelCount = width * height;
    randStates = createObjectArray<hiprandState>(pixelCount);

    renderInit<<<gridSize, blockSize>>>(width, height, randStates);
    gpuErrorCheck(hipDeviceSynchronize());

    imageData = std::make_shared<ImageData>();

    imageData->width = width;
    imageData->height = height;
    imageData->channels = 3;
    imageData->size = pixelCount * 3;

    Utils::reportGPUUsageInfo();
}   

void clearBackBuffers() {
    clearBackBuffers<<<gridSize, blockSize>>>(canvas);
    gpuErrorCheck(hipDeviceSynchronize());
    canvas->resetSampleCount();
    canvas->resetRenderingTime();
}

void pathTracing() {
#ifdef GPU_REALTIME
    if (camera->isDirty()) {
        clearBackBuffers();
        camera->updateViewMatrix();
        camera->resetDiryFlag();
    }

    canvas->incrementSampleCount();
    canvas->incrementRenderingTime(frameTime * 1000.0f);
    renderKernel<<<gridSize, blockSize>>>(canvas, camera, randStates, nullptr);
    gpuErrorCheck(hipDeviceSynchronize());

    imageData->data = canvas->getPixelBuffer();
#else
    auto* counter = createObjectPtr<int32_t>();

    canvas->incrementSampleCount();
    renderKernel<<<gridSize, blockSize>>>(canvas, camera, randStates, counter);
    gpuErrorCheck(hipDeviceSynchronize());

    deleteObject(counter);
#endif
}

void cleanup() {
    deleteObject(randStates);

    deleteDeviceObjectArray<<<1, 1>>>(triangles, triangleCount);
    deleteDeviceObjectArray<<<1, 1>>>(spheres, OBJECTS);
    deleteDeviceObjectArray<<<1, 1>>>(materials, MATERIALS);

    gpuErrorCheck(hipDeviceSynchronize());

    deleteObject(triangles);
    deleteObject(spheres);
    deleteObject(materials);

    deleteObject(camera);
    canvas->uninitialize();
    deleteObject(canvas);
}

#ifndef GPU_REALTIME
int main() {
    gpuErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 8192));

    initialize(width, height);
    
    GPUTimer timer("Rendering start...");
    pathTracing();
    timer.stop("Rendering elapsed time");

    canvas->writeToPNG("render.png");
    Utils::openImage(L"render.png");

    cleanup();

    return 0;
}
#endif // !GPU_REALTIME