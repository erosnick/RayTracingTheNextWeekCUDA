#include "hip/hip_runtime.h"
﻿
#include "main.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include "Utils.h"
#include "GPUTimer.h"
#include "Sphere.h"
#include "Plane.h"
#include "YAML.h"

#include <yaml-cpp/yaml.h>
#include <cstdio>

template<typename T>
T* createObjectPtr() {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T*)));
    return object;
}

template<typename T>
T* createObjectArray(int32_t numObjects) {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T) * numObjects));
    return object;
}

template<typename T>
T* createObjectPtrArray(int32_t numObjects) {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T) * numObjects));
    return object;
}

template<typename T>
void deleteObject(T* object) {
    gpuErrorCheck(hipFree(object));
}

template<typename T>
CUDA_GLOBAL void deleteDeviceObject(T** object) {
    delete (*object);
}

template<typename T>
CUDA_GLOBAL void deleteDeviceObjectArray(T** object, int32_t count) {
    for (auto i = 0; i < count; i++) {
        delete *(object + i);
    }
}

constexpr auto BOUNCES = 4;

constexpr auto SPHERES = 8;
constexpr auto MATERIALS = 10;
CUDA_CONSTANT Hitable* constantSpheres[SPHERES];
CUDA_CONSTANT Material* constantMaterials[MATERIALS];

CUDA_DEVICE bool hit(const Ray& ray, Float tMin, Float tMax, HitResult& hitResult) {
    HitResult tempHitResult;
    bool bHitAnything = false;
    Float closestSoFar = tMax;
    for (auto& sphere : constantSpheres) {
        // Empty hit call costs ~130ms
        if (sphere->hit(ray, tMin, closestSoFar, tempHitResult)) {
            bHitAnything = true;
            closestSoFar = tempHitResult.t;
            hitResult = tempHitResult;
        }
    }

    return bHitAnything;
}

using ScatterFunction = bool (*)(const Ray& inRay, const HitResult& hitResult, Float3& attenuation, Ray& scattered, hiprandState* randState);

CUDA_DEVICE ScatterFunction scatterFunction;

CUDA_DEVICE bool lambertianScatter(const Ray& inRay, const HitResult& hitResult, Float3& attenuation, Ray& scattered, hiprandState* randState) {
    auto scatterDirection = hitResult.normal + Utils::randomUnitVector(randState);
    if (Utils::nearZero(scatterDirection)) {
        scatterDirection = hitResult.normal;
    }
    scattered = Ray(inRay.at(hitResult.t), normalize(scatterDirection), inRay.time);
    attenuation = make_float3(1.0f, 1.0f, 1.0f);
    return true;
}

CUDA_DEVICE bool emissionScatter(const Ray& inRay, const HitResult& hitResult, Float3& attenuation, Ray& scattered, hiprandState* randState) {
    attenuation = make_float3(1.0f, 1.0f, 1.0f) * 5.0f;
    return false;
}

CUDA_DEVICE Float3 rayColor(const Ray& ray, hiprandState* randState) {
    Ray currentRay = ray;
    auto currentAttenuation = make_float3(1.0f, 1.0f, 1.0f);
    for (auto i = 0; i < BOUNCES; i++) {
        HitResult hitResult;
        // Smaller tMin will has a impact on performance
        if (hit(currentRay, Math::epsilon, Math::infinity, hitResult)) {
            Float3 attenuation;
            Ray scattered;
            // Bounces 4 Samples 100 18ms
            // Bounces 4 Samples 100 33ms(Empty scatter function body)
            if (constantMaterials[hitResult.materialId]->scatter(currentRay, hitResult, attenuation, scattered, randState)) {
            //if (lambertianScatter(currentRay, hitResult, attenuation, scattered, randState)) {
                currentAttenuation *= attenuation;
                currentRay = scattered;
            }
            else {
                return currentAttenuation * attenuation;
            }
        }
        // If no intersection in the first bounce, just return background color
        // otherwise return currentAttenuation * background color
        else {
            auto unitDirection = normalize(currentRay.direction);
            auto t = 0.5f * (unitDirection.y + 1.0f);
            auto background = lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), t);
            return currentAttenuation * background;
        }
    }
    // exceeded recursion
    return make_float3(0.0f, 0.0f, 0.0f);
}

//CUDA_DEVICE Float3 rayColor(const Ray& ray, hiprandState* randState, Sphere* spheres, int32_t depth) {
//    if (depth == 0) {
//        // exceeded recursion
//        return make_float3(0.0f, 0.0f, 0.0f);
//    }
//    HitResult hitResult;
//    // Smaller tMin will has a impact on performance
//    if (hit(ray, Math::epsilon, Math::infinity, hitResult, spheres)) {
//        Float3 attenuation;
//        Ray rayScattered;
//        if (hitResult.material->scatter(ray, hitResult, attenuation, rayScattered, randState)) {
//            return attenuation * rayColor(rayScattered, randState, spheres, depth - 1);
//        }
//        else {
//            return currentAttenuation * attenuation;
//        }
//    }
//
//    auto unitDirection = normalize(ray.direction);
//    auto t = 0.5f * (unitDirection.y + 1.0f);
//    auto background = lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), t);
//    return background;
//}

CUDA_GLOBAL void renderInit(int32_t width, int32_t height, hiprandState* randState) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto index = y * width + x;

    if (index < (width * height)) {
        //Each thread gets same seed, a different sequence number, no offset
        hiprand_init(1984, index, 0, &randState[index]);
    }
}

//CUDA_GLOBAL void render(Canvas canvas, Camera camera, hiprandState* randStates, Sphere* spheres) {
//    auto x = threadIdx.x + blockDim.x * blockIdx.x;
//    auto y = threadIdx.y + blockDim.y * blockIdx.y;
//    auto width = canvas.getWidth();
//    auto height = canvas.getHeight();
//    constexpr auto samplesPerPixel = 1;
//    constexpr auto maxDepth = 5;
//    auto index = y * width + x;
//
//    if (index < (width * height)) {
//        auto color = make_float3(0.0f, 0.0f, 0.0f);
//        auto localRandState = randStates[index];
//        for (auto i = 0; i < samplesPerPixel; i++) {
//
//            auto rx = hiprand_uniform(&localRandState);
//            auto ry = hiprand_uniform(&localRandState);
//
//            auto dx = Float(x + rx) / (width - 1);
//            auto dy = Float(y + ry) / (height - 1);
//
//            auto ray = camera.getRay(dx, dy);
//            color += rayColor(ray, &localRandState, spheres);
//        }
//        // Very important!!!
//        randStates[index] = localRandState;
//        canvas.writePixel(index, color / samplesPerPixel);
//    }
//}

CUDA_GLOBAL void renderKernel(Canvas* canvas, Camera* camera, hiprandState* randStates, int32_t* counter) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto width = canvas->getWidth();
    auto height = canvas->getHeight();
#ifdef GPU_REALTIME
    constexpr auto samplesPerPixel = 1;
#else
    constexpr auto samplesPerPixel = 1024;
#endif // GPU_REALTIME

    constexpr auto maxDepth = 5;
    auto index = y * width + x;

    if (index < (width * height)) {
        auto color = make_float3(0.0f, 0.0f, 0.0f);
        auto localRandState = randStates[index];
        for (auto i = 0; i < samplesPerPixel; i++) {

            auto rx = hiprand_uniform(&localRandState);
            auto ry = hiprand_uniform(&localRandState);

            auto dx = Float(x + rx) / (width - 1);
            auto dy = Float(y + ry) / (height - 1);

            auto ray = camera->getRay(dx, dy, &localRandState);
            color += rayColor(ray, &localRandState);
        }
        // Very important!!!
        randStates[index] = localRandState; 
#ifdef GPU_REALTIME
        canvas->accumulatePixel(index, color);
#else
        canvas->writePixel(index, color / samplesPerPixel);

        auto tenPercent = (width * height) / 10;

        auto old = atomicAdd(counter, 1);

        if ((old + 1) > 0 && (old + 1) % tenPercent == 0) {
            printf("%.2f%%\n", (float((old + 1) * 100) / (width * height)));
        }
#endif // GPU_REALTIME
    }
}

CUDA_GLOBAL void createLambertianMaterialKernel(Material** material, int32_t index, Float3 albedo, Float absorb = 1.0f) {
    *(material + index) = new Lambertian(index, albedo, absorb);
}

void createLambertianMaterial(Material** material, int32_t index, Float3 albedo, Float absorb = 1.0f) {
    createLambertianMaterialKernel<<<1, 1>>>(material, index, albedo, absorb);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createEmissionMaterialKernel(Material** material, int32_t index, Float3 albedo, Float intensity = 1.0f) {
    *(material + index) = new Emission(index, albedo, intensity);
}

void createEmissionMaterial(Material** material, int32_t index, Float3 albedo, Float intensity = 1.0f) {
    createEmissionMaterialKernel<<<1, 1>>>(material, index, albedo, intensity);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createMetalMaterialKernel(Material** material, int32_t index, Float3 albedo, Float fuzz = 1.0f) {
    *(material + index) = new Metal(index, albedo, fuzz);
}

void createMetalMaterial(Material** material, int32_t index, Float3 albedo, Float fuzz = 1.0f) {
    createMetalMaterialKernel<<<1, 1>>>(material, index, albedo, fuzz);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createDieletricMaterialKernel(Material** material, int32_t index, Float indexOfRefraction = 1.5f) {
    *(material + index) = new Dieletric(index, indexOfRefraction);
}

void createDieletricMaterial(Material** material, int32_t index, Float indexOfRefraction = 1.5f) {
    createDieletricMaterialKernel<<<1, 1>>>(material, index, indexOfRefraction);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void clearBackBuffers(Canvas* canvas) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto width = canvas->getWidth();
    auto height = canvas->getHeight();

    auto index = y * width + x;

    if (index < (width * height)) {
        canvas->clearPixel(index);
    }
}

CUDA_GLOBAL void createSphereKernel(Hitable** sphere, int32_t index, Float3 center, Float radius, Material* material, bool bShading) {
    *(sphere + index) = new Sphere(center, radius, material, bShading);
}

void createSphere(Hitable** sphere, int32_t index, Float3 center, Float radius, Material* material, bool bShading = true) {
    createSphereKernel<<<1, 1>>>(sphere, index, center, radius, material, bShading);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createMovingSphereKernel(Hitable** sphere, int32_t index, Float3 center0, Float3 center1, Float time0, Float time1, Float radius, Material* material) {
    *(sphere + index) = new MovingSphere(center0, center1, time0, time1, radius, material);
}

void createMovingSphere(Hitable** sphere, int32_t index, Float3 center0, Float3 center1, Float time0, Float time1, Float radius, Material* material) {
    createMovingSphereKernel<<<1, 1>>>(sphere, index, center0, center1, time0, time1, radius, material);
    gpuErrorCheck(hipDeviceSynchronize());
}

CUDA_GLOBAL void createPlaneKernel(Hitable** plane , int32_t index, Float3 position, Float3 normal, Float3 extend, Material* material) {
    *(plane + index) = new Plane(position, normal, extend, material);
}

void createPlane(Hitable** plane, int32_t index, const Float3& position, const Float3& normal, const Float3& extend, Material* material) {
    createPlaneKernel<<<1, 1>>>(plane, index, position, normal, extend, material);
    gpuErrorCheck(hipDeviceSynchronize());
}

#define RESOLUTION 1

#if RESOLUTION == 0
int32_t width = 512;
int32_t height = 384;
#elif RESOLUTION == 1
int32_t width = 512;
int32_t height = 512;
#elif RESOLUTION == 2
int32_t width = 1024;
int32_t height = 576;
#elif RESOLUTION == 3
int32_t width = 1280;
int32_t height = 720;
#elif RESOLUTION == 4
int32_t width = 1920;
int32_t height = 1080;
#elif RESOLUTION == 5
int32_t width = 64;
int32_t height = 36;
#endif

#define SCENE 1

Canvas* canvas = nullptr;
Camera* camera = nullptr;
Hitable** spheres = nullptr;
//std::vector<Material**> materials(MATERIALS);
Material** materials = nullptr;
hiprandState* randStates = nullptr;
std::shared_ptr<ImageData> imageData = nullptr;

dim3 blockSize(32, 32);
dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
              (height + blockSize.y - 1) / blockSize.y);

void loadScene(const std::string& path) {
    YAML::Node config = YAML::LoadFile(path);

    printf("name:%s\n", config["name"].as<std::string>().c_str());
    printf("sex:%s\n", config["sex"].as<std::string>().c_str());
    printf("age:%d\n", config["age"].as<int>());

    for (auto iterator = config["skills"].begin(); iterator != config["skills"].end(); iterator++) {
        printf("%s\n", iterator->first.as<std::string>().c_str());
    }
}

void initialize(int32_t width, int32_t height) {
    //Canvas canvas(width, height);
    Utils::reportGPUUsageInfo();
    canvas = createObjectPtr<Canvas>();
    canvas->initialize(width, height);

    //Camera camera(make_float3(-2.0f, 2.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 20.0f);
    camera = createObjectPtr<Camera>();
    //camera->initialize(make_float3(-2.0f, 2.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 20.0f);
    //camera->initialize(make_float3(0.0f, 1.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 90.0f);

    //auto eye = make_float3(3.0f, 3.0f, 5.0f);
    //auto center = make_float3(0.0f, 0.0f, -1.0f);
    //auto up = make_float3(0.0f, 1.0f, 0.0f);
    //auto focusDistance = length(center - eye);
    //camera->initialize(eye, center, up, Float(width) / height, 20.0f, 2.0f, focusDistance);

    // If the distance between object and camera equals to focus lens
    // then the object is in focus
    //auto eye = position(3.0f, 3.0f, 5.0f);
    //auto center = position(0.0f, 0.0f, -1.0f);
    //auto up = position(0.0f, 1.0f, 0.0f);
    //auto focusDistance = length(center - eye);
    //camera->initialize(eye, center, up, Float(width) / height, 20.0f, 2.0f, focusDistance);

    //loadScene("./resources/scenes/test.yaml");

    //for (auto& material : materials) {
    //    material = createObjectPtr<Material*>();
    //}

    materials = createObjectPtrArray<Material*>(MATERIALS);

    spheres = createObjectPtrArray<Hitable*>(SPHERES);

#if SCENE == 0
    // If the distance between object and camera equals to focus lens
    // then the object is in focus
    auto eye = point(3.0f, 3.0f, 5.0f);
    auto center = point(0.0f, 0.0f, -1.0f);
    auto up = point(0.0f, 1.0f, 0.0f);
    auto focusDistance = length(center - eye);
    auto aperture = 0.0f;
    camera->initialize(eye, center, up, Float(width) / height, 20.0f, aperture, focusDistance, 0.0f, 1.0f);

    // Scene1 Defocus Blur
    createDieletricMaterial(materials[0], 1.5f);
    createDieletricMaterial(materials[1], 1.5f);
    createLambertianMaterial(materials[2], make_float3(0.1f, 0.2f, 0.5f));
    //createDieletricMaterial<<<1, 1>>>(materials[3], 1.5f);
    createMetalMaterial(materials[3], make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    //createLambertianMaterial<<<1, 1>>>(materials[4], make_float3(0.8f, 0.8f, 0.0f));
    createMetalMaterial(materials[4], make_float3(0.5f, 0.7f, 1.0f), 0.0f);

    auto center1 = point(0.0f, 0.5f, 0.0f);

    //createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f}, 0.5f, *(materials[0]));
    //createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    //createSphere(spheres, 2, {  0.0f, 0.0f, -1.0f },  0.5f, *(materials[2]));
    //createSphere(spheres, 3, {  1.0f, 0.0f, -1.0f },  0.5f, *(materials[3]));
    //createSphere(spheres, 4, {  0.0f, -100.5f, -1.0f }, 100.0f, *(materials[4]));
    createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f}, 0.5f, *(materials[0]));
    createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    createMovingSphere(spheres, 2, {  0.0f, 0.0f, -1.0f }, { 0.0f, 0.5f, -1.0f }, 0.0f, 1.0f, 0.5f, *(materials[2]));
    createSphere(spheres, 3, {  1.0f, 0.0f, -1.0f },  0.5f, *(materials[3]));
    createSphere(spheres, 4, {  0.0f, -100.5f, -1.0f }, 100.0f, *(materials[4]));
#elif SCENE == 1
    // If the distance between object and camera equals to focus lens
    // then the object is in focus
    auto eye = point(0.0f, 0.0f, 1.25f);
    auto center = point(0.0f, 0.0f, -1.0f);
    auto up = point(0.0f, 1.0f, 0.0f);
    auto focusDistance = length(center - eye);
    auto aperture = 0.0f;
    camera->initialize(eye, center, up, Float(width) / height, 60.0f, aperture, focusDistance, 0.0f, 1.0f);

    //// Scene1 Defocus Blur
    ////createLambertianMaterial(materials[0], make_float3(0.0f, 1.0f, 0.0f));
    ////createLambertianMaterial(materials[1], make_float3(1.0f, 0.0f, 0.0f));
    //createLambertianMaterial(materials, 2, make_float3(1.0f, 1.0f, 1.0f));
    createDieletricMaterial(materials, 3, 1.5f);
    createMetalMaterial(materials, 4, make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    createLambertianMaterial(materials, 5, make_float3(0.8f, 0.8f, 0.0f));
    createMetalMaterial(materials, 6, make_float3(0.5f, 0.7f, 1.0f), 0.0f);

    ////createLambertianMaterial(materials[5], make_float3(1.0f, 1.0f, 1.0f));
    createDieletricMaterial(materials, 7, 1.5f);
    createDieletricMaterial(materials, 8, 1.5f);
    createEmissionMaterial(materials, 9, make_float3(1.0f, 1.0f, 1.0f), 15.0f);

    YAML::Node scene = YAML::LoadFile("./resources/scenes/scene.yaml");

    auto objects = scene["objects"];

    for (auto i = 0; i < SPHERES - 1; i++) {
        auto object = objects[i];

        auto center = object["sphere"]["center"].as<Float3>();
        auto radius = object["sphere"]["radius"].as<Float>();
        auto materialId = object["sphere"]["materialId"].as<uint32_t>();
        auto materialType = static_cast<MaterialType>(object["sphere"]["material"]["type"].as<uint8_t>());

        switch (materialType) {
            case MaterialType::Lambertian: {
                auto albedo = object["sphere"]["material"]["albedo"].as<Float3>();
                if ((materials[materialId]) == nullptr) {
                    createLambertianMaterial(materials, materialId, albedo);
                }
            }
        
            break;
        default:
            break;
        }
        
        createSphere(spheres, i, center, radius, materials[materialId]);
    }

    createPlane(spheres, SPHERES - 1, { 0.0f, 0.495f, 0.0f }, { 0.0f, 1.0f, 0.0f }, { 0.125f, 0.125f, 0.125f }, materials[9]);

    //// If the distance between object and camera equals to focus lens
    //// then the object is in focus
    //auto eye = point(0.0f, 0.0f, 1.5f);
    //auto center = point(0.0f, 0.0f, -1.0f);
    //auto up = point(0.0f, 1.0f, 0.0f);
    //auto focusDistance = length(center - eye);
    //auto aperture = 0.0f;
    //camera->initialize(eye, center, up, Float(width) / height, 45.0f, aperture, focusDistance, 0.0f, 1.0f);

    //// Scene1 Defocus Blur
    //createDieletricMaterial(materials[0], 1.5f);
    //createDieletricMaterial(materials[1], 1.5f);
    //createLambertianMaterial(materials[2], make_float3(0.1f, 0.2f, 0.5f));
    ////createDieletricMaterial<<<1, 1>>>(materials[3], 1.5f);
    //createMetalMaterial(materials[3], make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    ////createLambertianMaterial<<<1, 1>>>(materials[4], make_float3(0.8f, 0.8f, 0.0f));
    //createMetalMaterial(materials[4], make_float3(0.5f, 0.7f, 1.0f), 0.0f);
    //createLambertianMaterial(materials, 5, make_float3(1.0f, 1.0f, 1.0f));

    ////createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f}, 0.5f, *(materials[0]));
    ////createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    ////createSphere(spheres, 2, {  0.0f, 0.0f, -1.0f },  0.5f, *(materials[2]));
    ////createSphere(spheres, 3, {  1.0f, 0.0f, -1.0f },  0.5f, *(materials[3]));
    ////createSphere(spheres, 4, {  0.0f, -100.5f, -1.0f }, 100.0f, *(materials[4]));
    ////createSphere(spheres, 0, { -1.0f, 0.0f, -1.0f }, 0.5f, *(materials[5]));
    ////createSphere(spheres, 1, { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false);
    ////createSphere(spheres, 2, { 0.0f, 0.0f, -1.0f }, 0.5f, *(materials[5]));
    //createSphere(spheres, 0, {  0.25f, -0.325f, -0.125f }, 0.175f, *(materials[5]));
    //createSphere(spheres, 1, { -0.25f, -0.325f, -0.25f }, 0.175f, * (materials[5]));
    //createSphere(spheres, 2, {  0.0f, -1000.5f, 0.0f }, 1000.0f, *(materials[5]));
#else
    auto eye = point(13.0f, 2.0f, 3.0f);
    auto center = point(0.0f, 0.0f, 0.0f);
    auto up = point(0.0f, 1.0f, 0.0f);
    auto focusDistance = 10.0f;
    auto aperture = 0.1f;
    camera->initialize(eye, center, up, Float(width) / height, 20.0f, aperture, focusDistance, 0.0f, 1.0f);

    // Scene2 Final
    for (auto a = -11; a < 11; a++) {
        for (auto b = -11; b < 11; b++) {
            auto index = (a + 11) * 22 + (b + 11);
            auto chooseMaterial = Utils::randomFloat();

            auto center = point(a + 0.9f * Utils::randomFloat(), 0.2f, b + 0.9f * Utils::randomFloat());

            if (length(center - point(4.0f, 0.2f, 0.0f)) > 0.9f) {
                if (chooseMaterial < 0.8f) {
                    // Diffuse
                    auto albedo = Color::random() * Color::random();
                    createLambertianMaterial(materials[index], albedo);
                    auto center1 = center + point(0.0f, Utils::randomFloat(0.0f, 0.5f), 0.0f);
                    createMovingSphere(spheres, index, center, center1, 0.0f, 1.0f, 0.2f, *(materials[index]));
                    //createSphere(spheres, index, center, 0.2f, *(materials[index]));

                }
                else if (chooseMaterial < 0.95f) {
                    // Metal
                    auto albedo = Color::random(0.5f, 1.0f);
                    auto fuzz = Utils::randomFloat(0.0f, 0.5f);
                    createMetalMaterial(materials[index], albedo, fuzz);
                    createSphere(spheres, index, center, 0.2f, *(materials[index]));
                }
                else {
                    // Glass
                    createDieletricMaterial(materials[index], 1.5f);
                    createSphere(spheres, index, center, 0.2f, *(materials[index]));
                }
            }
            else {
                auto albedo = Color::random() * Color::random();
                createLambertianMaterial(materials[index], albedo);
                createSphere(spheres, index, center, 0.2f, *(materials[index]));
            }
        }
    }

    createLambertianMaterial(materials[484], color(0.5f, 0.5f, 0.5f));
    createDieletricMaterial(materials[485], 1.5f);
    createLambertianMaterial(materials[486], color(0.4f, 0.2f, 0.1f), 1.0f);
    createMetalMaterial(materials[487], color(0.7f, 0.6f, 0.5f), 0.0f);

    createSphere(spheres, 484, point( 0.0f, -1000.0,  0.0f), 1000.0f, *(materials[484]));
    createSphere(spheres, 485, point( 0.0f,     1.0f, 0.0f),    1.0f, *(materials[485]));
    createSphere(spheres, 486, point(-4.0f,     1.0f, 0.0f),    1.0f, *(materials[486]));
    createSphere(spheres, 487, point( 4.0f,     1.0f, 0.0f),    1.0f, *(materials[487]));

#endif
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(constantSpheres), spheres, sizeof(Hitable*) * SPHERES));
    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(constantMaterials), materials, sizeof(Material*) * MATERIALS));

    auto pixelCount = width * height;
    randStates = createObjectArray<hiprandState>(pixelCount);

    renderInit<<<gridSize, blockSize>>>(width, height, randStates);
    gpuErrorCheck(hipDeviceSynchronize());

    imageData = std::make_shared<ImageData>();

    imageData->width = width;
    imageData->height = height;
    imageData->channels = 3;
    imageData->size = pixelCount * 3;

    Utils::reportGPUUsageInfo();
}   

void clearBackBuffers() {
    clearBackBuffers<<<gridSize, blockSize>>>(canvas);
    gpuErrorCheck(hipDeviceSynchronize());
    canvas->resetSampleCount();
    canvas->resetRenderingTime();
}

void pathTracing() {
#ifdef GPU_REALTIME
    if (camera->isDirty()) {
        clearBackBuffers();
        camera->updateViewMatrix();
        camera->resetDiryFlag();
    }

    canvas->incrementSampleCount();
    canvas->incrementRenderingTime(frameTime * 1000.0f);
    renderKernel<<<gridSize, blockSize>>>(canvas, camera, randStates, nullptr);
    gpuErrorCheck(hipDeviceSynchronize());

    imageData->data = canvas->getPixelBuffer();
#else
    auto* counter = createObjectPtr<int32_t>();

    canvas->incrementSampleCount();
    renderKernel<<<gridSize, blockSize>>>(canvas, camera, randStates, counter);
    gpuErrorCheck(hipDeviceSynchronize());

    deleteObject(counter);
#endif
}

void cleanup() {
    deleteObject(randStates);

    deleteDeviceObjectArray<<<1, 1>>>(spheres, SPHERES);
    deleteDeviceObjectArray<<<1, 1>>>(materials, MATERIALS);

    gpuErrorCheck(hipDeviceSynchronize());

    deleteObject(spheres);
    deleteObject(materials);

    deleteObject(camera);
    canvas->uninitialize();
    deleteObject(canvas);
}

#ifndef GPU_REALTIME
int main() {
    gpuErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 8192));

    initialize(width, height);
    
    GPUTimer timer("Rendering start...");
    pathTracing();
    timer.stop("Rendering elapsed time");

    canvas->writeToPNG("render.png");
    Utils::openImage(L"render.png");

    cleanup();

    return 0;
}
#endif // !GPU_REALTIME