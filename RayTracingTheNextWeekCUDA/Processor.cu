#include "hip/hip_runtime.h"
#include "Processor.h"

/** Macro for checking if CUDA has problems */
#define cudaCheckError() { \
    hipError_t err = hipGetLastError(); \
    if(err != hipSuccess) { \
      printf("Cuda error: %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
      exit(1); \
    } \
  }

/**
 * A simple image processing kernel that copies the inverted data from the input surface to the output surface.
 */
__global__ void kernel(hipSurfaceObject_t input, int width, int height, uint8_t* data) {

    //Get the pixel index
    unsigned int xPx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int yPx = threadIdx.y + blockIdx.y * blockDim.y;

    auto index = yPx * width + xPx;
    //Don't do any computation if this thread is outside of the surface bounds.
    if (index >= (width * height)) return;

    uchar4 pixel;
    pixel.x = data[index * 3 + 0];
    pixel.y = data[index * 3 + 1];
    pixel.z = data[index * 3 + 2];
    pixel.w = 255;
    surf2Dwrite(pixel, input, xPx * sizeof(uchar4), yPx);
}

void Processor::setInput(uint8_t* const data, int imageWidth, int imageHeight)
{
    //Same-size images don't need texture regeneration, so skip that.
    if (imageHeight == imageInputDimensions.height && imageWidth == imageInputDimensions.width) {


        /*
        Possible shortcut: we know the input is the same size as the texture and CUDA surface object.
        So instead of destroying the surface and texture, why not just overwrite them?

        That's what I try to do in the following block, but because "data" is BGR and the texture
        is RGBA, the channels get all messed up.
        */

        //Use the input surface's CUDAResourceDesc to gain access to the surface data array
#ifdef USE_1
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        cudaGetSurfaceObjectResourceDesc(&resDesc, d_imageInputTexture);
        cudaCheckError();
        uint8_t* data4 = new uint8_t[imageInputDimensions.width * imageInputDimensions.height * 4];
        for (int i = 0; i < imageInputDimensions.width * imageInputDimensions.height; i++) {
            data4[i * 4 + 0] = data[i * 3 + 0];
            data4[i * 4 + 1] = data[i * 3 + 1];
            data4[i * 4 + 2] = data[i * 3 + 2];
            data4[i * 4 + 3] = 255;
        }
        //Copy the data from the input array to the surface
//        hipMemcpyToArray(resDesc.res.array.array, 0, 0, data, imageInputDimensions.width * imageInputDimensions.height * 3, hipMemcpyHostToDevice);
        hipMemcpy2DToArray(resDesc.res.array.array, 0, 0, data4, imageInputDimensions.width * 4, imageInputDimensions.width * 4, imageInputDimensions.height, hipMemcpyHostToDevice);
        cudaCheckError();
        delete[] data4;
#endif
        //Set status flags
        surfacesInitialized = true;

        return;
    }


    //Clear everything that originally existed in the texture/surface
    destroyEverything();

    //Get the size of the image and place it here.
    imageInputDimensions.width = imageWidth;
    imageInputDimensions.height = imageHeight;
    imageOutputDimensions.width = imageWidth;
    imageOutputDimensions.height = imageHeight;

    //Create the input surface/texture pair
    createTextureSurfacePair(imageInputDimensions, data, imageInputTexture, d_imageInputGraphicsResource, d_imageInputTexture);

    //Set status flags
    surfacesInitialized = true;
}

void Processor::processData(uint8_t* data)
{
    //Call the algorithm

    //Set the number of blocks to call the kernel with.
    dim3 blockSize(32, 32);
    dim3 gridSize((imageInputDimensions.width + blockSize.x - 1) / blockSize.x,
                  (imageInputDimensions.height + blockSize.y - 1) / blockSize.y);

    kernel<<<gridSize, blockSize >>>(d_imageInputTexture, imageInputDimensions.width, imageInputDimensions.height, data);

    //Sync the surface with the texture
    hipDeviceSynchronize();
    cudaCheckError();
}

GLuint Processor::getInputTexture()
{
    return imageInputTexture;
}

void Processor::writeOutputTo(uint8_t* destination)
{
    //Haven't figured this out yet
}

void Processor::createTextureSurfacePair(const Processor::ImgDim& dimensions, uint8_t* const data, GLuint& textureOut, hipGraphicsResource_t& graphicsResourceOut, hipSurfaceObject_t& surfaceOut) {

    // Create the OpenGL texture that will be displayed with GLAD and GLFW
    glGenTextures(1, &textureOut);
    // Bind to our texture handle
    glBindTexture(GL_TEXTURE_2D, textureOut);
    // Set texture interpolation methods for minification and magnification
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // Set texture clamping method
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    // Create the texture and its attributes
    glTexImage2D(GL_TEXTURE_2D,     // Type of texture
        0,                // Pyramid level (for mip-mapping) - 0 is the top level
        GL_RGB,          // Internal color format to convert to
        dimensions.width,            // Image width  i.e. 640 for Kinect in standard mode
        dimensions.height,           // Image height i.e. 480 for Kinect in standard mode
        0,                // Border width in pixels (can either be 1 or 0)
        GL_RGB,          // Input image format (i.e. GL_RGB, GL_RGBA, GL_BGR etc.)
        GL_UNSIGNED_BYTE, // Image data type.
        data);            // The actual image data itself
    //Note that the type of this texture is an RGBA UNSIGNED_BYTE type. When CUDA surfaces
    //are synchronized with OpenGL textures, the surfaces will be of the same type.
    //They won't know or care about their data types though, for they are all just byte arrays
    //at heart. So be careful to ensure that any CUDA kernel that handles a CUDA surface
    //uses it as an appropriate type. You will see that the update_surface kernel (defined 
    //above) treats each pixel as four unsigned bytes along the X-axis: one for red, green, blue,
    //and alpha respectively.

    //Create the CUDA array and texture reference
    hipArray* bitmap_d;
    //Register the GL texture with the CUDA graphics library. A new hipGraphicsResource is created, and its address is placed in cudaTextureID.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__OPENGL.html#group__CUDART__OPENGL_1g80d12187ae7590807c7676697d9fe03d
    hipGraphicsGLRegisterImage(&graphicsResourceOut, textureOut, GL_TEXTURE_2D,
        hipGraphicsRegisterFlagsNone);
    cudaCheckError();
    //Map graphics resources for access by CUDA.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1gad8fbe74d02adefb8e7efb4971ee6322
    hipGraphicsMapResources(1, &graphicsResourceOut, 0);
    cudaCheckError();
    //Get the location of the array of pixels that was mapped by the previous function and place that address in bitmap_d
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1g0dd6b5f024dfdcff5c28a08ef9958031
    hipGraphicsSubResourceGetMappedArray(&bitmap_d, graphicsResourceOut, 0, 0);
    cudaCheckError();
    //Create a CUDA resource descriptor. This is used to get and set attributes of CUDA resources.
    //This one will tell CUDA how we want the bitmap_surface to be configured.
    //Documentation for the struct: https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaResourceDesc.html#structcudaResourceDesc
    struct hipResourceDesc resDesc;
    //Clear it with 0s so that some flags aren't arbitrarily left at 1s
    memset(&resDesc, 0, sizeof(resDesc));
    //Set the resource type to be an array for convenient processing in the CUDA kernel.
    //List of resTypes: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g067b774c0e639817a00a972c8e2c203c
    resDesc.resType = hipResourceTypeArray;
    //Bind the new descriptor with the bitmap created earlier.
    resDesc.res.array.array = bitmap_d;
    //Create a new CUDA surface ID reference.
    //This is really just an unsigned long long.
    //Docuentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1gbe57cf2ccbe7f9d696f18808dd634c0a
    surfaceOut = 0;
    //Create the surface with the given description. That surface ID is placed in bitmap_surface.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__SURFACE__OBJECT.html#group__CUDART__SURFACE__OBJECT_1g958899474ab2c5f40d233b524d6c5a01
    hipCreateSurfaceObject(&surfaceOut, &resDesc);
    cudaCheckError();
}

void Processor::destroyEverything()
{
    if (surfacesInitialized) {

        //Input image CUDA surface
        hipDestroySurfaceObject(d_imageInputTexture);
        hipGraphicsUnmapResources(1, &d_imageInputGraphicsResource);
        hipGraphicsUnregisterResource(d_imageInputGraphicsResource);
        d_imageInputTexture = 0;

        //Input image GL texture
        glDeleteTextures(1, &imageInputTexture);
        imageInputTexture = 0;

        surfacesInitialized = false;
    }
}
