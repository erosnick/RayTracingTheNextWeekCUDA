#include "hip/hip_runtime.h"
#include "Sphere.h"

CUDA_DEVICE bool Sphere::hit(const Ray& ray, Float tMin, Float tMax, HitResult& hitResult) const {
    auto oc = ray.origin - center;
    auto a = dot(ray.direction, ray.direction);
    auto halfB = dot(oc, ray.direction);
    auto c = dot(oc, oc) - radius * radius;
    auto discriminant = halfB * halfB - a * c;
    // Cant's use Math::epsilon(0.001f) for comparison here
    // Because it's not small enough(Not precise enough)
    auto bHit = (discriminant > 0.0f);

    if (!bHit) {
        return false;
    }

    auto sqrtd = sqrt(discriminant);
    Float root = (-halfB - sqrtd) / a;

    // Find the nearest root that lies in the acceptable range.
    if (root < tMin || tMax < root) {
        root = (-halfB + sqrtd) / a;
        if (root < tMin || tMax < root) {
            return false;
        }
    }

    hitResult.bHit = true;
    hitResult.t = root;
    hitResult.position = ray.at(hitResult.t);
    auto outwardNormal = (hitResult.position - center) / radius;
    hitResult.setFaceNormal(ray, outwardNormal);
    hitResult.material = material;
    return true;
}

CUDA_DEVICE bool MovingSphere::hit(const Ray& ray, Float tMin, Float tMax, HitResult& hitResult) const {
    auto oc = ray.origin - center(ray.time);
    auto a = dot(ray.direction, ray.direction);
    auto halfB = dot(oc, ray.direction);
    auto c = dot(oc, oc) - radius * radius;
    auto discriminant = halfB * halfB - a * c;
    // Cant's use Math::epsilon(0.001f) for comparison here
    // Because it's not small enough(Not precise enough)
    auto bHit = (discriminant > 0.0f);

    if (!bHit) {
        return false;
    }

    auto sqrtd = sqrt(discriminant);
    Float root = (-halfB - sqrtd) / a;

    // Find the nearest root that lies in the acceptable range.
    if (root < tMin || tMax < root) {
        root = (-halfB + sqrtd) / a;
        if (root < tMin || tMax < root) {
            return false;
        }
    }

    hitResult.bHit = true;
    hitResult.t = root;
    hitResult.position = ray.at(hitResult.t);
    auto outwardNormal = (hitResult.position - center(ray.time)) / radius;
    hitResult.setFaceNormal(ray, outwardNormal);
    hitResult.material = material;
    return true;
}

CUDA_DEVICE Float3 MovingSphere::center(Float time) const {
    auto newCenter = center0 + ((time - time0) / (time1 - time0)) * (center1 - center0);
    return newCenter;
}